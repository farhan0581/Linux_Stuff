/*
WAP in Cuda C/C++, display resultant sum of array 
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void myAdder(int * array, int *sum,int N)

{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
		*sum	=	*sum + array[idx];
}

int main()
{
	int i,*sum_h,*sum_d,*array_h,*array_d,N,size,n_block,block_size;
	i=0;
	sum_h=&i;
	printf("Enter size of array\n");
	scanf("%d",&N);
	size = sizeof(int)*N;

	// allocating memory in host
	array_h = (int *)malloc(size);

	// allocating memory in device
	hipMalloc((void **)&array_d, size);
	hipMalloc((void **)&sum_d, sizeof(int));
	for(i=0;i<N;i++)
	{
		printf("enter element %d::",i+1);
		scanf("%d",&array_h[i]);
	}
	// copy data from  host memory to device memory 
	hipMemcpy(array_d,array_h,size,hipMemcpyHostToDevice);
	hipMemcpy(sum_d,sum_h,sizeof(int),hipMemcpyHostToDevice);

	// Number of thread per block
	printf("Enter block size (Number of thread per block):: ");
	scanf("%d",&block_size);

	// number of block
	n_block = N/block_size + (N%block_size == 0 ? 0:1);

	// Calling  device function
	myAdder<<< n_block,block_size >>>(array_d,sum_d,N-1);

	// copy data from device memory to host memory
	hipMemcpy(sum_h,sum_d,sizeof(int),hipMemcpyDeviceToHost);

	printf("Final Answer is %d\n",*sum_h);
	return 0;
}