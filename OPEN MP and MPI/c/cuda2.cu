/* Write a program in cuda c/c++ to add two number and display result*/
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void add(int *x,int *y,int *z)
{
	*z = *x + *y;
}
int main()
{
	int a,b,c;
	int *d_a,*d_b,*d_c;
	
	hipMalloc((void **)&d_a,sizeof(int));
	hipMalloc((void **)&d_b,sizeof(int));
	hipMalloc((void **)&d_c,sizeof(int));
	printf("enter first number\n");
	scanf("%d",&a);
	printf("enter second number\n");
	scanf("%d",&b);
	hipMemcpy(d_a,&a,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_b,&b,sizeof(int),hipMemcpyHostToDevice);
	add<<<1,1>>>(d_a,d_b,d_c);
	hipMemcpy(&c,d_c,sizeof(int),hipMemcpyDeviceToHost);
	
	printf("result is %d\n",c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return(0);
}
